#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void dot_product_kernel(const float* d_a, const float* d_b, float* d_result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Каждый поток обрабатывает одну пару элементов
    if (idx < n) {
        atomicAdd(d_result, d_a[idx] * d_b[idx]);  // Атомарно обновляем результат
    }
}

int main() {
    int n = 1024;  // Размер векторов
    std::vector<float> a(n, 1.0f);  // Вектор a, заполняем единицами
    std::vector<float> b(n, 2.0f);  // Вектор b, заполняем двойками

    float* d_a;
    float* d_b;
    float* d_result;
    float h_result = 0.0f;

    // Выделяем память на устройстве
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_result, sizeof(float));

    // Копируем данные с хоста на устройство
    hipMemcpy(d_a, a.data(), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    // Запускаем ядро CUDA с 256 потоками в блоке
    int block_size = 256;
    int num_blocks = (n + block_size - 1) / block_size;
    dot_product_kernel<<<num_blocks, block_size>>>(d_a, d_b, d_result, n);

    // Ждем завершения вычислений
    hipDeviceSynchronize();

    // Копируем результат с устройства на хост
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Выводим результат
    std::cout << "Скалярное произведение: " << h_result << std::endl;

    // Освобождаем память
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
