#include <iostream>
#include <hip/hip_runtime.h>

__device__ bool check_condition(int row, int col) {
    // Пример условной функции
    return (row + col) % 2 == 0;  // Условие: если сумма индексов чётная
}

__global__ void count_threads(int* res, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows && col < cols) {
        if (check_condition(row, col)) {
            atomicAdd(res, 1);  // Атомарное увеличение глобальной переменной
        }
    }
}

int main() {
    const int rows = 1024;  // Размер сетки
    const int cols = 1024;
    
    int* d_res;
    int h_res = 0;

    // Выделяем память на устройстве для переменной res
    hipMalloc(&d_res, sizeof(int));
    hipMemcpy(d_res, &h_res, sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);  // Размер блока
    dim3 numBlocks((cols + 15) / 16, (rows + 15) / 16);  // Количество блоков

    // Запускаем ядро
    count_threads<<<numBlocks, threadsPerBlock>>>(d_res, rows, cols);
    hipDeviceSynchronize();

    // Копируем результат обратно на хост
    hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);

    // Выводим результат
    std::cout << "Количество нитей, удовлетворяющих условию: " << h_res << std::endl;

    // Освобождаем память
    hipFree(d_res);

    return 0;
}
